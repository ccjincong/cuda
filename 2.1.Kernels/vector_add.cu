
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void VecAdd(float* A, float* B, float* C)
{
	    int i = threadIdx.x;
	        C[i] = A[i] + B[i];
}

int main()
{
	const int N = 32; //长度
	float A[N],B[N],C[N];

	for(int i=0;i<N;i++)
	{
		A[i]=i;
		B[i]=i;
	}

	// Allocate memory on the device
	float *dev_A, *dev_B, *dev_C;
	hipMalloc(&dev_A, N * sizeof(float));
	hipMalloc(&dev_B, N * sizeof(float));
	hipMalloc(&dev_C, N * sizeof(float));

	// Copy input vectors to device
	hipMemcpy(dev_A, A, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, N * sizeof(float), hipMemcpyHostToDevice);
	// Launch kernel with N threads
	VecAdd<<<1, N>>>(dev_A, dev_B, dev_C);

	// Copy output vector from device
	hipMemcpy(C, dev_C, N * sizeof(float), hipMemcpyDeviceToHost);

	// Free memory on the device
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	// Print output vector
	for (int i = 0; i < N; i++)
	{
		printf("%f ", C[i]);
	}
	printf("\n");

	return 0;
}




